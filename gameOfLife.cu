// Author @ Eric Reinsmidt
// Date @ 2014.11.23
// Version 0.1

/*

  Hi Eduardo, I've made an assumption in the code. I don't set the device using cudaSetDevice().
  So, on a multi-GPU system this code will default to device 0, which is whatever
  device is in the first slot. So please keep that in mind if testing on a multi-GPU
  system.

  I did not assume a finite grid.

  I've prepopulated the automaton with a row of 10 cells. This leads to the creation of
  a pentadecathlon oscillator:
  http://www.conwaylife.com/wiki/Pentadecathlon

  Also if you are bored, I coded the Game of Life four or five years ago in JavaScript
  that is fun to play with:
  http://reinsmidt.com/snippets/life/

*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
using namespace std;

void outputAutomaton(char *automaton) {
  int cellNum = 0;
  for(int i = 0; i < 256; i++) {
    for(int j = 0; j < 256; j++) {
      if(automaton[cellNum] == 1) {
        cout << "@";
      } else {
        cout << " ";
      }
      cellNum++;
    }
    cout << endl;
  }
  return;
}

// Translate from index into threads to rows and colums
__device__ void translateToRowAndCol(int index, int *row, int *col, int rows, int cols) {
  *row = index / rows;
  *col = index % cols;
  return;
}

// Translate from rows and columns to index into threads
__device__ void translateToIndex(int row, int col, int *index, int rows, int cols) {
  *index = __umul24(row, cols) + col;
}

// Check neighbor's health and update cell accordingly
__global__ void changeCellState(char *currGen, char *nextGen, int rows, int cols) {
  
  // Calculate index into array
  int index = __umul24(blockDim.x, blockIdx.x) + threadIdx.x;
  
  int colIndex;
  int rowIndex;
  int newIndex;

  translateToRowAndCol(index, &rowIndex, &colIndex, rows, cols);
  translateToIndex(rowIndex, colIndex, &newIndex, rows, cols);

  int cellNeighbors = 0;
  int tempIndex;

  //////////////////////////////////
  // Normal cases around neighbor //
  //////////////////////////////////

  // Check upper neighbor
  if (rowIndex != 0) {
    translateToIndex((rowIndex - 1), colIndex, &tempIndex, rows, cols);
    if (currGen[tempIndex] == 1) {
      cellNeighbors++;
    }
  }

  // Check upper right neighbor
  if (rowIndex != 0 && colIndex != cols - 1) {
    translateToIndex((rowIndex - 1), (colIndex + 1), &tempIndex, rows, cols);
    if (currGen[tempIndex] == 1) {
      cellNeighbors++;
    }
  }

  // Check right neighbor
  if (colIndex != cols - 1) {
    translateToIndex(rowIndex, (colIndex + 1), &tempIndex, rows, cols);
    if(currGen[tempIndex] == 1) {
      cellNeighbors++;
    }
  }

  // Check bottom right neighbor
  if (colIndex != cols - 1 && rowIndex != rows - 1) {
    translateToIndex((rowIndex + 1), (colIndex + 1), &tempIndex, rows, cols);
    if (currGen[tempIndex] == 1) {
      cellNeighbors++;
    }
  }

  // Check bottom neighbor
  if (rowIndex != rows - 1) {
    translateToIndex((rowIndex + 1), colIndex, &tempIndex, rows, cols);
    if (currGen[tempIndex] == 1) {
      cellNeighbors++;
    }
  }

  // Check bottom left neighbor
  if (rowIndex != rows - 1 && colIndex != 0) {
    translateToIndex((rowIndex + 1), (colIndex - 1), &tempIndex, rows, cols);
    if (currGen[tempIndex] == 1) {
      cellNeighbors++;
    }
  }

  // Check left neighbor
  if (colIndex != 0) {
    translateToIndex(rowIndex, (colIndex - 1), &tempIndex, rows, cols);
    if (currGen[tempIndex] == 1) {
      cellNeighbors++;
    }
  }

  // Check upper left neighbor
  if (colIndex != 0 && rowIndex != 0) {
    translateToIndex((rowIndex - 1), (colIndex - 1), &tempIndex, rows, cols);
    if (currGen[tempIndex] == 1) {
      cellNeighbors++;
    }
  }

  //////////////////////////////////////////
  // Special cases like edges and corners //
  //////////////////////////////////////////
  
  // Upper row, wrap to bottom row
  if (rowIndex == 0) {
    translateToIndex((rows - 1), colIndex, &tempIndex, rows, cols);
    if (currGen[tempIndex]) {
      cellNeighbors++;
    }
  }

  // Bottom row, wrap to top row
  if (rowIndex == (rows - 1)) {
    translateToIndex(0, colIndex, &tempIndex, rows, cols);
    if (currGen[tempIndex]) {
      cellNeighbors++;
    }
  }

  // Left column, wrap to right column
  if (colIndex == 0) {
    translateToIndex(rowIndex, (cols - 1), &tempIndex, rows, cols);
    if (currGen[tempIndex]) {
      cellNeighbors++;
    }
  }

  // Right column, wrap to left column
  if (colIndex == (cols - 1)) {
    translateToIndex(rowIndex, 0, &tempIndex, rows, cols);
    if (currGen[tempIndex]) {
      cellNeighbors++;
    }
  }

  // Upper left, wrap to bottom right
  if (rowIndex == 0 && colIndex == 0) {
    translateToIndex((rows - 1), (cols - 1), &tempIndex, rows, cols);
    if (currGen[tempIndex]) {
      cellNeighbors++;
    }
  }

  // Upper right, wrap to bottom left
  if (rowIndex == 0 && colIndex == (cols - 1)) {
    translateToIndex((rows - 1), 0, &tempIndex, rows, cols);
    if (currGen[tempIndex]) {
      cellNeighbors++;
    }
  }

  // Bottom right, wrap to upper left
  if (rowIndex == (rows - 1) && colIndex == (cols - 1)) {
    translateToIndex(0, 0, &tempIndex, rows, cols);
    if (currGen[tempIndex]) {
      cellNeighbors++;
    }
  }

  // Bottom left, wrap to upper right
  if (rowIndex == (rows - 1) && colIndex == 0) {
    translateToIndex(0, (cols - 1), &tempIndex, rows, cols);
    if (currGen[tempIndex]) {
      cellNeighbors++;
    }
  }

  //__syncthreads();

  // Determine if cell lives, dies, or is born by evaluating how many neighbors it has
  if (currGen[index] == 1) { // Live cell
    if (cellNeighbors < 2 || cellNeighbors > 3) {
      nextGen[index] = 0; // Died from underpopulation or overcrowding
    } else {
      nextGen[index] = 1; // Still alive
    }
  } else { // Dead cell
    if (cellNeighbors == 3) {
      nextGen[index] = 1; // Born
    } else {
      nextGen[index] = 0; // Still dead
    }
  }
}