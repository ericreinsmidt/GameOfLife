// Author @ Eric Reinsmidt
// Date @ 2014.11.23
// Version 0.1

/*

  Driver cuda program for gameOfLife.cu

*/

#include <stdio.h>
#include <iostream>
#include "gameOfLife.cu"
using namespace std;

int main() {

    // Flag to output all generations to stdout
    bool showAll = false;

    // /Init number of generations to run
    int numGenerations = 100;

    // Create arrays for current and next generation on host
    char currentGeneration[65536];
    char theNextGeneration[65536];

    // Create pointers to current and next generation on device
    char *currentGenerationOnDevice;
    char *theNextGenerationOnDevice;

    // Fill automaton with empty cells
    for (int i = 0; i < 65536; i++) {
      currentGeneration[i] = 0;
    }

    // Place a strip of 10 vertical cells in middle of automaton
    // This is what will become the pentadecathlon oscillator
    for (int i = 123; i < 133; i++) {
      currentGeneration[i * 256 + 128] = 1;
    }

    // Output starting generation to stdout if flag set
    if (showAll) {
      cout << "Starting Generation:" << endl;
      outputAutomaton(currentGeneration);
    }

    // Allocate memory for current generation on device
    if (hipMalloc((void **)&currentGenerationOnDevice, 65536 * sizeof(char)) != hipSuccess) {
      cout << "hipMalloc() failed!" << endl;
      exit(0);
    }

    // Allocate memory for the next generation on device
    if (hipMalloc((void **)&theNextGenerationOnDevice, 65536 * sizeof(char)) != hipSuccess) {
      cout << "hipMalloc() failed!" << endl;
      exit(0);
    }

    // Copy initial generation from host to device
    if (hipMemcpy(currentGenerationOnDevice, currentGeneration, 65536 * sizeof(char), hipMemcpyHostToDevice) != hipSuccess) {
      cout << "hipMemcpy() failed!" << endl;
      exit(0);
    }

    // Continue calculating next generation until desired number of generations
    for(int i = 0; i < numGenerations; i++) {
      
      changeCellState <<<64, 1024>>>(currentGenerationOnDevice, theNextGenerationOnDevice, 256, 256);

      // Block until the device has completed all preceding requested tasks
      if (hipDeviceSynchronize() != hipSuccess) {
        cout << "hipDeviceSynchronize() failed!" << endl;
        exit(0);
      }

      // Output current generation to stdout if flag set
      if (showAll) {
        hipMemcpy(currentGeneration, theNextGenerationOnDevice, 65536 * sizeof(char), hipMemcpyDeviceToHost);
        outputAutomaton(currentGeneration);
      }

      // Copy calculated generation to current generation on device
      hipMemcpy(currentGenerationOnDevice, theNextGenerationOnDevice, 65536 * sizeof(char), hipMemcpyDeviceToDevice);
    }

    // Copy final generation on device to host
    hipMemcpy(theNextGeneration, theNextGenerationOnDevice, 65536 * sizeof(char), hipMemcpyDeviceToHost);
    
    // Output final generation to stdout
    cout << "Final Generation:" << endl;
    outputAutomaton(theNextGeneration);

    // Explicitly destroy and clean up all resources associated
    // with the current device in the current process.
    if (hipDeviceReset() != hipSuccess) {
        cout << "hipDeviceReset() failed!" << endl;
        return 1;
    }

    return 0;
}